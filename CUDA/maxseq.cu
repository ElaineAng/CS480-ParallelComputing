#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <math.h>

long getmax(long *, long);

__global__ void getmaxcu(long * num, long size, long * max_ans){
  
  __device__ __shared__ long local_max_num;
  __device__ __shared__ long local_arr[1024];
    
  local_max_num = 10;
  local_arr[threadIdx.x] = num[blockDim.x * blockIdx.x + threadIdx.x];
  __syncthreads();

  int i;
  for (i=0; i<blockDim.x; i++){
      if (threadIdx.x == i && (local_arr[i] > local_max_num) && (local_arr[i] < size)){
	 local_max_num = local_arr[i];
      }
  }
  
  if (threadIdx.x == 0){
     	atomicMax((unsigned long long *) max_ans, 
  		(unsigned long long) local_max_num);
  }
   
}

int main(int argc, char *argv[]){
   long size = 0;  // The size of the array
   long i;  // loop index
   long * numbers; //pointer to the array
    
    if(argc !=2)
    {
       printf("usage: maxseq num\n");
       printf("num = size of the array\n");
       exit(1);
    }
   
    size = atol(argv[1]);

    numbers = (long *)malloc(size * sizeof(long));
    if( !numbers )
    {
       printf("Unable to allocate mem for an array of size %ld\n", size);
       exit(1);
    }    

    srand(time(NULL)); // setting a seed for the random number generator
    // Fill-up the array with random numbers from 0 to size-1 
    for( i = 0; i < size; i++)
       numbers[i] = rand() % size;    
   
    long * d_num;
    long * d_max;
    long h_max;
    h_max = 10;

    hipMalloc(&d_num, size * sizeof(long));
    hipMalloc(&d_max, sizeof(long));
    hipMemcpy(d_num, numbers, size*sizeof(long), hipMemcpyHostToDevice);
    hipMemcpy(d_max, &h_max, sizeof(long), hipMemcpyHostToDevice);

    hipDeviceProp_t prop;
    int device;
    
    hipGetDevice(&device);
    hipGetDeviceProperties(&prop, device);

    int thrdPerBlk = 1024;
    int blkPerGrid = (int) ceil((float) size / thrdPerBlk);

    getmaxcu <<<blkPerGrid, thrdPerBlk>>> (d_num, size, d_max);
    hipDeviceSynchronize();
    hipMemcpy(&h_max, d_max, sizeof(long), hipMemcpyDeviceToHost);
    

    printf("The maximum number from GPU is %ld.\n", h_max);
    
    hipFree(d_num);
    hipFree(d_max);
    free(numbers);
    exit(0);
}


/*
   input: pointer to an array of long int
          number of elements in the array
   output: the maximum number of the array

long getmax(long num[], long size)
{
  long i;
  long max = num[0];

  for(i = 1; i < size; i++)
	if(num[i] > max)
	   max = num[i];

  return( max );

}
*/
